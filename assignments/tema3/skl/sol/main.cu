#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include "helper.h"

using namespace std;

__global__ void accessiblePopulation(const int n, float *lats, float *lons,
                                    int *pops, int *res, const float kmRange) {
    register unsigned int index = threadIdx.x + blockDim.x * blockIdx.x;
    register float crt_lat = lats[index];
    register float crt_lon = lons[index];
    register int crt_pop = pops[index];

    for (register int i = index + 1; i < n; i++) {
        float phi1 = (90.f - crt_lat) * DEGREE_TO_RADIANS;
        float phi2 = (90.f - lats[i]) * DEGREE_TO_RADIANS;

        float theta1 = crt_lon * DEGREE_TO_RADIANS;
        float theta2 = lons[i] * DEGREE_TO_RADIANS;

        float cs = sin(phi1) * sin(phi2) * cos(theta1 - theta2) +
                    cos(phi1) * cos(phi2);
        if (cs > 1) {
            cs = 1;
        } else if (cs < -1) {
            cs = -1;
        }

        float dist = 6371.f * acos(cs);

        if (dist <= kmRange) {
            // res[index] += pops[i];
            // res[i] += crt_pop;
            atomicAdd(&res[index], pops[i]);
            atomicAdd(&res[i], crt_pop);
        }
    }
}

int countLines(const char* fileIn)
{
    register int cnt = 0;
    register string line;

    register ifstream ifs(fileIn);

    while(getline(ifs, line)) {
        cnt++;
    }

    ifs.close();

    return cnt;
}

void compute(float kmRange, const char* fileIn, const char* fileOut) {
    string geon;

    float *host_lats = 0;
    float *host_lons = 0;
    int *host_pops = 0;
    int *host_res = 0;

    float *device_lats = 0;
    float *device_lons = 0;
    int *device_pops = 0;
    int *device_res = 0;

    int n = countLines(fileIn);
    int i = 0;

    if (n > 200000)
        return;

    ifstream ifs(fileIn);
    ofstream ofs(fileOut);

    host_lats = (float *) malloc(n * sizeof(float));
    host_lons = (float *) malloc(n * sizeof(float));
    host_pops = (int *) malloc(n * sizeof(int));
    host_res = (int *) malloc(n * sizeof(int));

    hipMalloc((void **) &device_lats, n * sizeof(float));
    hipMalloc((void **) &device_lons, n * sizeof(float));
    hipMalloc((void **) &device_pops, n * sizeof(int));
    hipMalloc((void **) &device_res, n * sizeof(int));

    DIE(host_lats == 0 || host_lons == 0 ||
        host_pops == 0 || host_res == 0 ||
        device_lats == 0 || device_lons == 0 ||
        device_pops == 0 || device_res == 0,
        "malloc failed");

    while(ifs >> geon >> host_lats[i] >> host_lons[i] >> host_pops[i]) {
        host_res[i] = host_pops[i];
        i++;
    }

    n = i;

    hipMemcpy(device_lats, host_lats, n * sizeof(float),
                hipMemcpyHostToDevice);
    hipMemcpy(device_lons, host_lons, n * sizeof(float),
                hipMemcpyHostToDevice);
    hipMemcpy(device_pops, host_pops, n * sizeof(int),
                hipMemcpyHostToDevice);
    hipMemcpy(device_res, host_res, n * sizeof(int),
                hipMemcpyHostToDevice);

    int blockSize = 256;
    int numBlocks = n / blockSize;
    if (n % blockSize) {
        numBlocks++;
    }
    accessiblePopulation<<<numBlocks, blockSize>>>(n, device_lats, device_lons,
                                                    device_pops, device_res,
                                                    kmRange);

    hipDeviceSynchronize();

    hipMemcpy(host_res, device_res, n * sizeof(int),
    hipMemcpyDeviceToHost);

    for (i = 0; i < n; i++) {
        ofs << host_res[i] << endl;
    }

    free(host_lats);
    free(host_lons);
    free(host_pops);
    free(host_res);

    hipFree(device_lats);
    hipFree(device_lons);
    hipFree(device_pops);
    hipFree(device_res);

    ifs.close();
    ofs.close();
}

int main(int argc, char** argv) {
    DIE( argc == 1,
         "./accpop <kmrange1> <file1in> <file1out> ...");
    DIE( (argc - 1) % 3 != 0,
         "./accpop <kmrange1> <file1in> <file1out> ...");

    for(int argcID = 1; argcID < argc; argcID += 3) {
        float kmRange = atof(argv[argcID]);
        compute(kmRange, argv[argcID + 1], argv[argcID + 2]);
    }
}
