#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include "helper.h"

using namespace std;

__global__ void accessiblePopulation(int n, float *lats, float *lons,
                                    int *pops, int *res, float kmRange) {
    register unsigned int index = threadIdx.x + blockDim.x * blockIdx.x;
    register float crt_lat = lats[index];
    register float crt_lon = lons[index];
    register int crt_pop = pops[index];

    for (register int i = index + 1; i < n; i++) {
        float phi1 = (90.f - crt_lat) * DEGREE_TO_RADIANS;
        float phi2 = (90.f - lats[i]) * DEGREE_TO_RADIANS;

        float theta1 = crt_lon * DEGREE_TO_RADIANS;
        float theta2 = lons[i] * DEGREE_TO_RADIANS;

        float cs = sin(phi1) * sin(phi2) * cos(theta1 - theta2) +
                    cos(phi1) * cos(phi2);
        if (cs > 1) {
            cs = 1;
        } else if (cs < -1) {
            cs = -1;
        }

        float dist = 6371.f * acos(cs);

        if (dist <= kmRange) {
            // res[index] += pops[i];
            // res[i] += crt_pop;
            atomicAdd(&res[index], pops[i]);
            atomicAdd(&res[i], crt_pop);
        }
    }
}

int countLines(const char* fileIn)
{
    register int cnt = 0;
    register string line;

    register ifstream ifs(fileIn);

    while(getline(ifs, line)) {
        cnt++;
    }

    ifs.close();

    return cnt;
}

void compute(float kmRange, const char* fileIn, const char* fileOut) {
    string geon;
    float lat;
    float *lats;
    float lon;
    float *lons;
    int pop;
    int *pops;
    int *res;

    int n = countLines(fileIn);
    int i = 0;

    ifstream ifs(fileIn);
    ofstream ofs(fileOut);

    hipMallocManaged(&lats, n * sizeof(float));
    hipMallocManaged(&lons, n * sizeof(float));
    hipMallocManaged(&pops, n * sizeof(int));
    hipMallocManaged(&res, n * sizeof(int));

    while(ifs >> geon >> lat >> lon >> pop) {
        lats[i] = lat;
        lons[i] = lon;
        pops[i] = pop;
        res[i] = pop;
        i++;
    }

    n = i;

    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    accessiblePopulation<<<numBlocks, blockSize>>>(n, lats, lons, pops, res,
                                                    kmRange);

    hipDeviceSynchronize();

    for (i = 0; i < n; i++) {
        ofs << res[i] << endl;
    }

    hipFree(lats);
    hipFree(lons);
    hipFree(pops);
    hipFree(res);

    ifs.close();
    ofs.close();
}

int main(int argc, char** argv) {
    DIE( argc == 1,
         "./accpop <kmrange1> <file1in> <file1out> ...");
    DIE( (argc - 1) % 3 != 0,
         "./accpop <kmrange1> <file1in> <file1out> ...");

    for(int argcID = 1; argcID < argc; argcID += 3) {
        float kmRange = atof(argv[argcID]);
        compute(kmRange, argv[argcID + 1], argv[argcID + 2]);
    }
}
